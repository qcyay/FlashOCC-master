// Copyright (c) Phigent Robotics. All rights reserved.
// Reference https://arxiv.org/abs/2211.17111


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void nearest_assign_kernel(
                                  const int* l2s_key,
                                  int l2s_size,
                                  const int* occind2detind,
                                  const int *__restrict__ occ_pred,
                                  const int *__restrict__ inst_xyz,
                                  const int *__restrict__ inst_cls,
                                  const int *__restrict__ inst_id_list,
                                  int inst_size,
                                  int* __restrict__ inst_pred) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;  // 该pillar的cur_c特征对应的索引.
  int occ_pred_label = occ_pred[idx];
  int dist_min = 100000000;
  // int tgt_inst_id = -1;
  for (int index = 0; index < l2s_size; index ++)
  {
    if (occ_pred_label == l2s_key[index])
    {
      int x = idx/(200*16);
      int y = (idx - x*200*16)/16;
      int z = idx - x*200*16 - y*16;
      int inst_ind = 0;
      for (inst_ind = 0; inst_ind < inst_size; inst_ind ++)
      {
        if (inst_cls[inst_ind] == occind2detind[occ_pred_label])
        {
          int dx = x - inst_xyz[inst_ind*3+0];
          int dy = y - inst_xyz[inst_ind*3+1];
          int dz = z - inst_xyz[inst_ind*3+2];
          int dist = dx*dx + dy*dy + dz*dz;
          if (dist < dist_min){
            dist_min = dist;
            inst_pred[idx] = inst_id_list[inst_ind];
          }
        }
      }
      return;
    }
  }
  inst_pred[idx] = occ_pred[idx];
}

void nearest_assign(
              const int* l2s_key,
              int l2s_size,
              const int *__restrict__ occind2detind,
              int inst_size,
              const int *__restrict__ occ_pred,
              const int *__restrict__ inst_xyz,
              const int *__restrict__ inst_cls,
              const int *__restrict__ inst_id_list,
              int* __restrict__ inst_pred) {
  nearest_assign_kernel<<<(int)ceil(((double)200 * 200 * 16 / 256)), 256>>>(
    l2s_key, l2s_size, occind2detind, 
    occ_pred, inst_xyz, inst_cls, 
    inst_id_list, inst_size, inst_pred
  );
}


